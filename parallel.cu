#include "hip/hip_runtime.h"
/**
*   CUDA C/C++ implementation for Parallel Graph Coloring for Manycore Architectures
*   {@link https://ieeexplore.ieee.org/abstract/document/7516086}
*
*   @author Ashwin Joisa
*   @author Praveen Gupta
**/

//=============================================================================================//

// Include header files
#include <iostream>
#include <hip/hip_runtime.h>

// Include custom header file for implementation of Graphs
#include "Graph.h"

//=============================================================================================//

#define MAX_THREAD_COUNT 1024
#define CEIL(a, b) ((a - 1) / b + 1)

//=============================================================================================//

using namespace std;

float device_time_taken;

//=============================================================================================//

// Catch Cuda errors
void catchCudaError(hipError_t error, const char *function)
{
    if (error != hipSuccess)
    {
        printf("\n====== Cuda Error Code %i ======\n %s in CUDA %s\n", error, hipGetErrorString(error), function);
        exit(-1);
    }
}
//=============================================================================================//

__global__ void assignColoursKernel(Graph *graph, int nodeCount,
                                    int *device_colours, bool *device_conflicts, int maxDegree)
{

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= nodeCount || !device_conflicts[node])
        return;

    int maxColours = maxDegree + 1;
    // Create forbidden array of size maxDegree
    int *forbidden = new int[CEIL(maxColours + 1, 32)];
    memset(forbidden, 0, sizeof(int) * (maxColours + 1));

    for (int i = graph->adjacencyListPointers[node]; i < graph->adjacencyListPointers[node + 1]; i++)
    {
        int neighbour = graph->adjacencyList[i];
        int ind = device_colours[neighbour] % 32;
        forbidden[device_colours[neighbour] / 32] |= (1<<ind);
    }

    for (int colour = 1; colour <= maxColours; ++colour)
    {
        int ind = colour % 32;
        if ((forbidden[colour / 32] & (1<<ind)) == 0)
        {
            device_colours[node] = colour;
            break;
        }
    }

    delete[] forbidden;
}

void assignColours(Graph *graph, int nodeCount,
                   int *device_colours, bool *device_conflicts, int maxDegree)
{

    // Launch assignColoursKernel with nodeCount number of threads
    assignColoursKernel<<<CEIL(nodeCount, MAX_THREAD_COUNT), MAX_THREAD_COUNT>>>(graph, nodeCount, device_colours, device_conflicts, maxDegree);
    hipDeviceSynchronize();
}

__global__ void detectConflictsKernel(Graph *graph, int nodeCount,
                                      int *device_colours, bool *device_conflicts, bool *device_conflictExists)
{

    int node = blockIdx.x * blockDim.x + threadIdx.x;
    if (node >= nodeCount)
        return;

    device_conflicts[node] = false;

    for (int i = graph->adjacencyListPointers[node]; i < graph->adjacencyListPointers[node + 1]; i++)
    {
        int neighbour = graph->adjacencyList[i];
        if (device_colours[neighbour] == device_colours[node] && neighbour < node)
        {
            //conflict
            device_conflicts[node] = true;
            *device_conflictExists = true;
        }
    }
}

bool detectConflicts(Graph *graph, int nodeCount,
                     int *device_colours, bool *device_conflicts)
{

    bool *device_conflictExists;
    bool conflictExists = false;

    catchCudaError(hipMalloc((void **)&device_conflictExists, sizeof(bool)), "Malloc1");
    catchCudaError(hipMemcpy(device_conflictExists, &conflictExists, sizeof(bool), hipMemcpyHostToDevice), "Memcpy7");

    //Launch detectConflictsKernel with nodeCount number of threads
    detectConflictsKernel<<<CEIL(nodeCount, MAX_THREAD_COUNT), MAX_THREAD_COUNT>>>(graph, nodeCount, device_colours, device_conflicts, device_conflictExists);
    hipDeviceSynchronize();

    // Copy device_conflictExists to conflictExists and return
    catchCudaError(hipMemcpy(&conflictExists, device_conflictExists, sizeof(bool), hipMemcpyDeviceToHost), "Memcpy6");
    
    // Free device memory
    catchCudaError(hipFree(device_conflictExists), "Free");
    
    return conflictExists;
}

int *graphColouring(Graph *graph, int nodeCount, int maxDegree)
{

    // Boolean array for conflicts
    bool *host_conflicts = new bool[nodeCount];
    int *host_colours = new int[nodeCount];
    int *device_colours;
    bool *device_conflicts;

    // Initialize all nodes to invalid colour (0)
    memset(host_colours, 0, sizeof(int) * nodeCount);
    // Initialize all nodes into conflict
    memset(host_conflicts, true, sizeof(bool) * nodeCount);

    catchCudaError(hipMalloc((void **)&device_colours, sizeof(int) * nodeCount), "Malloc2");
    catchCudaError(hipMemcpy(device_colours, host_colours, sizeof(int) * nodeCount, hipMemcpyHostToDevice), "Memcpy1");
    catchCudaError(hipMalloc((void **)&device_conflicts, sizeof(bool) * nodeCount), "Malloc3");
    catchCudaError(hipMemcpy(device_conflicts, host_conflicts, sizeof(bool) * nodeCount, hipMemcpyHostToDevice), "Memcpy2");

    // Timer
    hipEvent_t device_start, device_end;
    catchCudaError(hipEventCreate(&device_start), "Event Create");
    catchCudaError(hipEventCreate(&device_end), "Event Create");
    catchCudaError(hipEventRecord(device_start), "Event Record");

    do
    {
        assignColours(graph, nodeCount, device_colours, device_conflicts, maxDegree);
    } while (detectConflicts(graph, nodeCount, device_colours, device_conflicts));

    // Timer
    catchCudaError(hipEventRecord(device_end), "Event Record");
    catchCudaError(hipEventSynchronize(device_end), "Event Synchronize");
    catchCudaError(hipEventElapsedTime(&device_time_taken, device_start, device_end), "Elapsed time");


    // Copy colours to host and return
    catchCudaError(hipMemcpy(host_colours, device_colours, sizeof(int) * nodeCount, hipMemcpyDeviceToHost), "Memcpy3");

    delete[] host_conflicts;
    catchCudaError(hipFree(device_colours), "Free");
    catchCudaError(hipFree(device_conflicts), "Free");

    return host_colours;
}

int main(int argc, char *argv[])
{

    if (argc < 2)
    {
        cout << "Usage: " << argv[0] << " <graph_input_file> [output_file]\n";
        return 0;
    }

    char choice;
    cout << "Would you like to print the colouring of the graph? (y/n) ";
    cin >> choice;

    freopen(argv[1], "r", stdin);

    Graph *host_graph = new Graph();
    Graph *device_graph;

    catchCudaError(hipMalloc((void **)&device_graph, sizeof(Graph)), "Malloc4");
    host_graph->readGraph();

    int nodeCount = host_graph->getNodeCount();
    int edgeCount = host_graph->getEdgeCount();
    int maxDegree = host_graph->getMaxDegree();
    catchCudaError(hipMemcpy(device_graph, host_graph, sizeof(Graph), hipMemcpyHostToDevice), "Memcpy4");

    // Copy Adjancency List to device
    int *adjacencyList;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyList, sizeof(int) * (2 * edgeCount + 1)), "Malloc5");
    catchCudaError(hipMemcpy(adjacencyList, host_graph->adjacencyList, sizeof(int) * (2 * edgeCount + 1), hipMemcpyHostToDevice), "Memcpy");
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyList), &adjacencyList, sizeof(int *), hipMemcpyHostToDevice), "Memcpy5");

    // Copy Adjancency List Pointers to device
    int *adjacencyListPointers;
    // Alocate device memory and copy
    catchCudaError(hipMalloc((void **)&adjacencyListPointers, sizeof(int) * (nodeCount + 1)), "Malloc6");
    catchCudaError(hipMemcpy(adjacencyListPointers, host_graph->adjacencyListPointers, sizeof(int) * (nodeCount + 1), hipMemcpyHostToDevice), "Memcpy");
    // Update the pointer to this, in device_graph
    catchCudaError(hipMemcpy(&(device_graph->adjacencyListPointers), &adjacencyListPointers, sizeof(int *), hipMemcpyHostToDevice), "Memcpy");


    int *colouring = graphColouring(device_graph, nodeCount, maxDegree);

    int chromaticNumber = INT_MIN;
    for (int i = 0; i < nodeCount; i++)
    {
        chromaticNumber = max(chromaticNumber, colouring[i]);
        if(choice == 'y' || choice == 'Y')
            printf("Node %d => Colour %d\n", i, colouring[i]);
    }
    cout << endl;
    printf("\nNumber of colours used (chromatic number) ==> %d\n", chromaticNumber);
    printf("Time Taken (Parallel) = %f ms\n", device_time_taken);

    if (argc == 3)
    {
        freopen(argv[2], "w", stdout);
        for (int i = 0; i < nodeCount; i++)
            cout << colouring[i] << " ";
        cout << endl;
    }

    // Free all memory
    delete[] colouring;
    catchCudaError(hipFree(adjacencyList), "Free");
    catchCudaError(hipFree(adjacencyListPointers), "Free");
    catchCudaError(hipFree(device_graph), "Free");
}